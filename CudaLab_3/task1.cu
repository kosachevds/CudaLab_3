#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <cstdlib>
#include <vector>
#include <chrono>
#include <algorithm>
#include <iostream>

const size_t SHARED_BLOCK_SIZE = 1024;

__global__ void reduceMin(unsigned const* inData, unsigned* outData)
{
    // TODO: as figure 3.8
    __shared__ unsigned shared [SHARED_BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //if (i + blockDim.x < blockDim.x && inData[i + blockDim.x] < inData[i]) {
    //    shared[tid] = inData[i + blockDim.x];
    //} else {
        shared[tid] = inData[i];
    //}
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (shared[tid + s] < shared[tid]) {
                shared[tid] = shared[tid + s];
            }
        }
        __syncthreads();
    }
    if (tid == 0) {
        outData[blockIdx.x] = shared[0];
    }
}

///////////////////////////////////////////////////////////////////////////////

unsigned getMinCpu(std::vector<unsigned> const& values, float* ms_out);
unsigned getMinGpu(std::vector<unsigned> const& values, float* ms_out);
void fillRandom(std::vector<unsigned>& values, size_t size);

void Task1()
{
    std::vector<unsigned> values;
    fillRandom(values, 8192 * SHARED_BLOCK_SIZE);
    float ms;
    auto min_cpu = getMinCpu(values, &ms);
    std::cout << "CPU: " << min_cpu << " " << ms << " ms." << std::endl;
    auto min_gpu = getMinGpu(values, &ms);
    std::cout << "GPU: " << min_gpu << " " << ms << " ms." << std::endl;
}

unsigned getMinCpu(std::vector<unsigned> const& values, float* ms_out)
{
    auto min_value = values.front();
    auto start = std::chrono::high_resolution_clock::now();
    //auto min_value = *std::min_element(values.begin(), values.end());
    //std::this_thread::sleep_for(std::chrono::microseconds(10LL));
    //for (auto value: values) {
    //    if (value < min_value) {
    //        min_value = value;
    //    }
    //}
    for (size_t i = 0; i < values.size(); ++i) {
        if (values[i] < min_value) {
            min_value = values[i];
        }
    }
    auto end = std::chrono::high_resolution_clock::now();
    auto ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
    if (ms_out != nullptr) {
        *ms_out = ns.count() / 1.0e6F;
    }
    return min_value;
}

unsigned getMinGpu(std::vector<unsigned> const& values, float* ms_out)
{
    auto gpu_values = thrust::device_vector<unsigned>(values);
    auto raw_gpu_values = thrust::raw_pointer_cast(gpu_values.data());

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    auto block_count = values.size() / SHARED_BLOCK_SIZE;
    auto out_gpu = thrust::device_vector<unsigned>(block_count);
    auto out_raw = thrust::raw_pointer_cast(out_gpu.data());

    hipEventRecord(start);
    reduceMin<<<block_count, SHARED_BLOCK_SIZE>>>
        (raw_gpu_values, out_raw);
    hipEventRecord(end);
    hipEventSynchronize(end);
    float ms1;
    hipEventElapsedTime(&ms1, start, end);

    auto out_cpu = thrust::host_vector<unsigned>(out_gpu);
    float ms2;
    auto min_value = getMinCpu({ out_cpu.begin(), out_cpu.end() }, &ms2);
    if (ms_out != nullptr) {
        *ms_out = ms1 + ms2;
    }

    hipEventDestroy(start);
    hipEventDestroy(end);
    std::cout << std::endl;
    return min_value;
}

void fillRandom(std::vector<unsigned>& values, size_t size)
{
    values.resize(size);
    auto add = size / (rand() % size + 1);
    for (size_t i = 0; i < size; ++i) {
        values[i] = rand() % (2 * size) + add;
    }
}
