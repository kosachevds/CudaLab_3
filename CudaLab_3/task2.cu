#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <cstdlib>
#include <vector>
#include <chrono>
#include <random>
#include <iostream>
#include <fstream>

const auto HISTOGRAM_SIZE = 256;

__global__ void histogramKernel( unsigned * result, unsigned * data, int n )
{

}

///////////////////////////////////////////////////////////////////////////////

void fillHistogramCpu(std::vector<unsigned> const& values, std::vector<unsigned>& histogram);
void fillWithNormalDistribution(std::vector<unsigned>& values, size_t size);
void writeVector(std::vector<unsigned> const& values, std::ostream& out);

void task2()
{
    auto values = std::vector<unsigned>();
    fillWithNormalDistribution(values, 1024 * 1024);
    auto histogram = std::vector<unsigned>();
    fillHistogramCpu(values, histogram);
    std::ofstream out("hist.txt");
    writeVector(histogram, out);
    out.close();
}

void fillHistogramCpu(std::vector<unsigned> const& values, std::vector<unsigned>& histogram)
{
    histogram.resize(HISTOGRAM_SIZE, 0u);
    for (auto item : values) {
        ++histogram[item];
    }
}

void fillWithNormalDistribution(std::vector<unsigned>& values, size_t size)
{
    values.resize(size);
    std::default_random_engine generator;
    std::normal_distribution<double> distribution(HISTOGRAM_SIZE / 2.0 + 1, HISTOGRAM_SIZE / 8.0);
    for (size_t i = 0; i < size; ++i) {
        auto value = -1;
        while (value < 0.0 || value >= HISTOGRAM_SIZE) {
            value = distribution(generator);
        }
        values[i] = static_cast<unsigned>(value);
    }
}

void writeVector(std::vector<unsigned> const& values, std::ostream& out)
{
    for (auto item : values) {
        out << item << " ";
    }
    out << std::endl;
}
